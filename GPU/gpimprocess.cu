#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>

#define TILE_W 16
#define TILE_H 16
#define R 1
#define D ((R*2)+1)
#define S (D*D)
#define BLOCK_W TILE_W
#define BLOCK_H TILE_H


//For debug purposes
void printImageMatrix(float** image, int height, int width)
{
  for(int i = 0; i < height; i++)
    {
      for(int j = 0; j < width; j++)
        {
          std::cout << image[i][j] << " ";
        }
      std::cout << "@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@" << std::endl;
    }
}


float* flattenImage(float** image, int height, int width)
{
   float* image_1D = new float[height*width];
   for(int i = 0; i < height; i++)
      {
         for(int j = 0; j < width; j++)
	    {
	       image_1D[j + i*width] = image[i][j];
	    }
      }
   std::cout << "Image flattened" << std::endl;
   return image_1D;
}


void fileToMatrix(float** &image, char* fileName, int* height, int* width)
{
   std::ifstream imageFile(fileName);

   if(imageFile.is_open())
      {
	std::string line;
	getline(imageFile, line);
	std::istringstream iss(line);
	iss >> *(height) >> *(width);
	std::cout << "Height: " << *height << " ||-|| Width: " << *width << std::endl;

	image = new float*[*(height)];

	for(int i = 0; i < *(height); i++)
	   {
              image[i] = new float[*(width)];
           }	  

      	int h = 0;
      	float val;
        while(getline(imageFile, line))
           {
              int w = 0;
              std::istringstream iss(line);
              while(iss >> val)
                 {
                    image[h][w++] = val;
                 }
              h++;
           }
        std::cout << "Image saved to matrix..." << std::endl;
      }

    imageFile.close();      
}

void matrixToFile(char* fileName, float* image, int height, int width)
{
  std::ofstream outFile;
  outFile.open(fileName);
  outFile << height << " " << width << '\n';
  for(int i = 0; i < height*width; i++)
    {
	int x = i % width;
	int y = i / width;
	if(i != 0 && x == 0)
	   outFile << '\n';
        outFile << ("%.2f", image[x + y*width]) << " ";
    }

  outFile.close();
}


void getMinAndMax(float** image, int height, int width, int* min, int* max)
{
   for(int i = 0; i < height; i++)
      {
         for(int j = 0; j < width; j++)
	    {
	       if(image[i][j] > *(max))
	          {
		     *max = image[i][j];
	          }
	       if(image[i][j] < *(min))
	          {
		     *min = image[i][j];
		  }
	    }
      }

   std::cout << "Min: " << *min << " ||-|| Max: " << *max << std::endl;
}

double getAverage(float** image, int height, int width)
{
	float sum = 0;

	for (int i = 0; i < height; i++)
	{
		for (int j = 0; j < width; j++)
		{
			sum += image[i][j];
		}
	}

	return sum / (width * height);
}


__global__ void linearScale(float* image, int* a, int* width, float* b)
{
   float u = image[blockIdx.x];
   image[blockIdx.x] = (*b) * (u + (*a));
}


__global__ void grayWorld(float* image, double* scalingValue)
{
	float u = image[blockIdx.x];
	image[blockIdx.x] = u * *scalingValue;
}

__global__ void reflection(float* image, int* width)
{
	//blockIdx.x = i * *width + j
	int i = blockIdx.x / *width;
	int j = blockIdx.x - i * *width;  

	int reflectedIndex = i * *width + (*width - j - 1);
	float u = image[reflectedIndex];
	image[blockIdx.x] = u;
}

__global__ void orderedDithering(float* image, int* width)
{
	//blockIdx.x = i * *width + j
	int i = blockIdx.x / *width;
	int j = blockIdx.x - i * *width; 

	float u = image[blockIdx.x];

	if(i%2 == 0)
	{
		if(j%2 == 0)
		{
			if(u > 192)
				image[blockIdx.x] = 256;
			else
				image[blockIdx.x] = 0;
		}
		else
		{
			if(u > 64)
				image[blockIdx.x] = 256;
			else
				image[blockIdx.x] = 0;
		}
	}
	else
	{
		if(j%2 == 0)
			image[blockIdx.x] = 256;

		else
		{
			if(u > 128)
				image[blockIdx.x] = 256;
			else
				image[blockIdx.x] = 0;
		}
	}

}

__global__ void rotate90(float* image, int* height, int* width)
{
	//blockIdx.x = i * *width + j
	int i = blockIdx.x / *width;
	int j = blockIdx.x - i * *width; 

	int rotatedIndex = (*height - j - 1) * *width + i;
	float u = image[rotatedIndex];
	image[blockIdx] = u;
}

__global__ void rotate180(float* image, int* height, int* width)
{
	//blockIdx.x = i * *width + j
	int i = blockIdx.x / *width;
	int j = blockIdx.x - i * *width; 

	int rotatedIndex = (*height - i - 1) * *width + (*width - j - 1);
	float u = image[rotatedIndex];
	image[blockIdx] = u;
}


__global__ void medianFilter(float* image, float* outImage, int* height, int* width)
{
   int x = blockIdx.x * TILE_W + threadIdx.x;
   int y = blockIdx.y * TILE_H + threadIdx.y;

   unsigned int index = y * (*width) + x;
   unsigned int block_index = (threadIdx.y * blockDim.y + threadIdx.x);
   
   __shared__ float sharedWindow[BLOCK_W * BLOCK_H];
   sharedWindow[block_index] = image[index];

   __syncthreads();

  int i, key, j;
  for (i = 1; i < block_index; i++)
     {
       key = sharedWindow[i];
       j = i - 1;

       while (j >= 0 && sharedWindow[j] > key)
         {
           sharedWindow[j + 1] = sharedWindow[j];
           j = j - 1;
         }
       sharedWindow[j + 1] = key;
     }

   outImage[index] = sharedWindow[block_index / 2];
}

int main(int argc, char** argv)
{
   char* inFileName = argv[1];
   char* outFileName = argv[2];
   int writeOut = atoi(argv[3]);


   //IMAGE INIT
   int height, width, max = -1, min = 256;
   float** hostImage;

   fileToMatrix(hostImage, inFileName, &height, &width);
   getMinAndMax(hostImage, height, width, &min, &max);
   float* hostFlattened = flattenImage(hostImage, height, width);  

   //GPU INIT
   const size_t image_size = sizeof(float) * size_t(height*width);

   float* hostResult = new float[height*width];
   float* deviceImage;
   int *height_d, *width_d;

   hipMalloc((void**)&height_d, sizeof(int));
   hipMalloc((void**)&width_d, sizeof(int));   

   hipMemcpy(height_d, &height, sizeof(int), hipMemcpyHostToDevice);         
   hipMemcpy(width_d, &width, sizeof(int), hipMemcpyHostToDevice);         


   //LINSCALE
   int BLOCK_C = height*width;
   int a = -1 * min, *d_a;

   float gmax = 255.0;
   float b = gmax / (max - min), *d_b;

   hipMalloc((void**)&d_a, sizeof(int));
   hipMalloc((void**)&d_b, sizeof(float));

   hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
   hipMemcpy(d_b, &b, sizeof(float), hipMemcpyHostToDevice);

   hipMalloc((void**)&deviceImage, image_size);
   hipMemcpy(deviceImage, hostFlattened, image_size, hipMemcpyHostToDevice);	

   linearScale<<<BLOCK_C, 1>>>(deviceImage, d_a, width_d, d_b);
   hipMemcpy(hostResult, deviceImage, image_size, hipMemcpyDeviceToHost);

   if(writeOut)
      matrixToFile(outFileName, hostResult, height, width);

   //GRAYWORLD
   double average = getAverage(hostImage, height, width), *d_average;


   hipMalloc((void**)&d_average, sizeof(double));
   hipMemcpy(d_average, &average, sizeof(double), hipMemcpyHostToDevice);

   hipMalloc((void**)&deviceImage, image_size);
   hipMemcpy(deviceImage, hostFlattened, image_size, hipMemcpyHostToDevice);	

   grayWorld<<<BLOCK_C, 1>>>(deviceImage, d_average);
   hipMemcpy(hostResult, deviceImage, image_size, hipMemcpyDeviceToHost);

   if(writeOut)
      matrixToFile(outFileName, hostResult, height, width);
   

  //REFLECTION
  
   hipMalloc((void**)&width_d, sizeof(int));          
   hipMemcpy(width_d, &width, sizeof(int), hipMemcpyHostToDevice);  

   hipMalloc((void**)&deviceImage, image_size);
   hipMemcpy(deviceImage, hostFlattened, image_size, hipMemcpyHostToDevice);	

   reflection<<<BLOCK_C, 1>>>(deviceImage, width_d);
   hipMemcpy(hostResult, deviceImage, image_size, hipMemcpyDeviceToHost);

   if(writeOut)
      matrixToFile(outFileName, hostResult, height, width);


   //ORDERED DITHERING
  
   hipMalloc((void**)&width_d, sizeof(int));          
   hipMemcpy(width_d, &width, sizeof(int), hipMemcpyHostToDevice);  

   hipMalloc((void**)&deviceImage, image_size);
   hipMemcpy(deviceImage, hostFlattened, image_size, hipMemcpyHostToDevice);	

   orderedDithering<<<BLOCK_C, 1>>>(deviceImage, width_d);
   hipMemcpy(hostResult, deviceImage, image_size, hipMemcpyDeviceToHost);

   if(writeOut)
      matrixToFile(outFileName, hostResult, height, width);


   //ROTATE180
  
   hipMalloc((void**)&height_d, sizeof(int));
   hipMalloc((void**)&width_d, sizeof(int));   

   hipMemcpy(height_d, &height, sizeof(int), hipMemcpyHostToDevice);         
   hipMemcpy(width_d, &width, sizeof(int), hipMemcpyHostToDevice);          

   hipMalloc((void**)&deviceImage, image_size);
   hipMemcpy(deviceImage, hostFlattened, image_size, hipMemcpyHostToDevice);	

   rotate180<<<BLOCK_C, 1>>>(deviceImage, height_d, width_d);
   hipMemcpy(hostResult, deviceImage, image_size, hipMemcpyDeviceToHost);

   if(writeOut)
      matrixToFile(outFileName, hostResult, height, width);


   /*MEDFILTER
   dim3 grid(TILE_W, TILE_H);
   dim3 block(BLOCK_W, BLOCK_H);

   float* outImage;
   hipMalloc((void**)&outImage, image_size);	

   hipMalloc((void**)&deviceImage, image_size);
   hipMemcpy(deviceImage, hostFlattened, image_size, hipMemcpyHostToDevice);

   medianFilter<<<grid, block>>>(deviceImage, outImage, height_d, width_d);
   hipMemcpy(hostResult, outImage, image_size, hipMemcpyDeviceToHost);

   if(writeOut)
      matrixToFile(outFileName, hostResult, height, width);*/

   return 0;
}
